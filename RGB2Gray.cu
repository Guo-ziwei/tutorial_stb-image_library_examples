#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#define CHANNELS 3

__global__ void colorConvert(unsigned char* grayImage, unsigned char* rgbImage, int width, int height) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    if (x < width && y < height) {
        int grayoffset = y * width + x;
        int rgboffset = grayoffset * CHANNELS;
        unsigned char r = rgbImage[rgboffset];
        unsigned char g = rgbImage[rgboffset + 1];
        unsigned char b = rgbImage[rgboffset + 2];
        grayImage[grayoffset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}

void rgb2gray(unsigned char* grayImage, unsigned char* rgbImage, int width, int height) {
    int grayoffset = 0, rgboffset = 0;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            grayoffset = y * width + x;
            rgboffset = grayoffset * CHANNELS;
            unsigned char r = rgbImage[rgboffset];
            unsigned char g = rgbImage[rgboffset + 1];
            unsigned char b = rgbImage[rgboffset + 2];
            grayImage[grayoffset] = 0.21f * r + 0.71f * g + 0.07f * b;
        }
    }
}

int main(int argc, char const* argv[]) {
    unsigned char* rgbimage;
    unsigned char* grayimage;
    int width, height, channels;
    struct timeval start, end;
    unsigned char* img = stbi_load("Shapes.png", &width, &height, &channels, CHANNELS);
    if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }
    printf("Loaded image with a width of %dpx, a height of %dpx and %d channels\n", width, height, CHANNELS);
    size_t img_size = width * height * CHANNELS;
    int gray_channels = CHANNELS == 4 ? 2 : 1;
    size_t gray_img_size = width * height * gray_channels;
    unsigned char* gray_img = (unsigned char*)malloc(gray_img_size);
    hipMalloc((void**)&rgbimage, img_size);
    hipMalloc((void**)&grayimage, gray_img_size);
    // Executing kernel
    dim3 block(width, 1, 1);
    dim3 grid(height, 1, 1);
    gettimeofday(&start, nullptr);
    hipMemcpy(rgbimage, img, img_size, hipMemcpyHostToDevice);
    hipMemcpy(grayimage, gray_img, gray_img_size, hipMemcpyHostToDevice);
    colorConvert<<<grid, block>>>(grayimage, rgbimage, width, height);
    hipMemcpy(gray_img, grayimage, gray_img_size, hipMemcpyDeviceToHost);
    gettimeofday(&end, nullptr);
    double elapsed_seconds = (end.tv_sec - start.tv_sec) * 1e3;
    elapsed_seconds += (end.tv_usec - start.tv_usec) * 1e-3;
    printf("gpu elapsed time: %lf\n", elapsed_seconds);
    // stbi_write_jpg("sky_gray.jpg", width, height, gray_channels, gray_img, 100);
    stbi_write_png("Shapes_gray_cuda.png", width, height, gray_channels, gray_img, width * gray_channels);
    printf("Write image with a width of %dpx, a height of %dpx and %d channels\n", width, height, gray_channels);
    gettimeofday(&start, nullptr);
    rgb2gray(gray_img, img, width, height);
    gettimeofday(&end, nullptr);
    elapsed_seconds = (end.tv_sec - start.tv_sec) * 1e3;
    elapsed_seconds += (end.tv_usec - start.tv_usec) * 1e-3;
    printf("elapsed time: %lf\n", elapsed_seconds);
    stbi_image_free(img);
    free(gray_img);
    hipFree(grayimage);
    hipFree(rgbimage);
    return 0;
}
